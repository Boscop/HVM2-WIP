#include "memory.cuh"

__host__ Net* mknet() {
  Net* net  = (Net*)malloc(sizeof(Net));
  net->root = mkptr(NIL, 0);
  net->rwts = 0;
  net->done = 0;
  net->blen = 0;
  net->bags = (Wire*)malloc(BAGS_SIZE * sizeof(Wire));
  net->gidx = (u32*) malloc(GIDX_SIZE * sizeof(u32));
  net->gtmp = (u32*) malloc(GTMP_SIZE * sizeof(u32));
  net->node = (Node*)malloc(NODE_SIZE * sizeof(Node));
  memset(net->bags, 0, BAGS_SIZE * sizeof(Wire));
  memset(net->gidx, 0, GIDX_SIZE * sizeof(u32));
  memset(net->gtmp, 0, GTMP_SIZE * sizeof(u32));
  memset(net->node, 0, NODE_SIZE * sizeof(Node));
  return net;
}

__host__ Net* net_to_device(Net* host_net) {
  // Allocate memory on the device for the Net object, and its data
  Net*  device_net;
  Wire* device_bags;
  u32*  device_gidx;
  u32*  device_gtmp;
  Node* device_node;

  hipMalloc((void**)&device_net, sizeof(Net));
  hipMalloc((void**)&device_bags, BAGS_SIZE * sizeof(Wire));
  hipMalloc((void**)&device_gidx, GIDX_SIZE * sizeof(u32));
  hipMalloc((void**)&device_gtmp, GTMP_SIZE * sizeof(u32));
  hipMalloc((void**)&device_node, NODE_SIZE * sizeof(Node));

  // Copy the host data to the device memory
  hipMemcpy(device_bags, host_net->bags, BAGS_SIZE * sizeof(Wire), hipMemcpyHostToDevice);
  hipMemcpy(device_gidx, host_net->gidx, GIDX_SIZE * sizeof(u32),  hipMemcpyHostToDevice);
  hipMemcpy(device_gtmp, host_net->gtmp, GTMP_SIZE * sizeof(u32),  hipMemcpyHostToDevice);
  hipMemcpy(device_node, host_net->node, NODE_SIZE * sizeof(Node), hipMemcpyHostToDevice);

  // Create a temporary host Net object with device pointers
  Net temp_net  = *host_net;
  temp_net.bags = device_bags;
  temp_net.gidx = device_gidx;
  temp_net.gtmp = device_gtmp;
  temp_net.node = device_node;

  // Copy the temporary host Net object to the device memory
  hipMemcpy(device_net, &temp_net, sizeof(Net), hipMemcpyHostToDevice);

  // Return the device pointer to the created Net object
  return device_net;
}

__host__ Net* net_to_host(Net* device_net) {
  // Create a new host Net object
  Net* host_net = (Net*)malloc(sizeof(Net));

  // Copy the device Net object to the host memory
  hipMemcpy(host_net, device_net, sizeof(Net), hipMemcpyDeviceToHost);

  // Allocate host memory for data
  host_net->bags = (Wire*)malloc(BAGS_SIZE * sizeof(Wire));
  host_net->gidx = (u32*) malloc(GIDX_SIZE * sizeof(u32));
  host_net->gtmp = (u32*) malloc(GTMP_SIZE * sizeof(u32));
  host_net->node = (Node*)malloc(NODE_SIZE * sizeof(Node));

  // Retrieve the device pointers for data
  Wire* device_bags;
  u32*  device_gidx;
  u32*  device_gtmp;
  Node* device_node;
  hipMemcpy(&device_bags, &(device_net->bags), sizeof(Wire*), hipMemcpyDeviceToHost);
  hipMemcpy(&device_gidx, &(device_net->gidx), sizeof(u32*),  hipMemcpyDeviceToHost);
  hipMemcpy(&device_gtmp, &(device_net->gtmp), sizeof(u32*),  hipMemcpyDeviceToHost);
  hipMemcpy(&device_node, &(device_net->node), sizeof(Node*), hipMemcpyDeviceToHost);

  // Copy the device data to the host memory
  hipMemcpy(host_net->bags, device_bags, BAGS_SIZE * sizeof(Wire), hipMemcpyDeviceToHost);
  hipMemcpy(host_net->gidx, device_gidx, GIDX_SIZE * sizeof(u32),  hipMemcpyDeviceToHost);
  hipMemcpy(host_net->gtmp, device_gtmp, GTMP_SIZE * sizeof(u32),  hipMemcpyDeviceToHost);
  hipMemcpy(host_net->node, device_node, NODE_SIZE * sizeof(Node), hipMemcpyDeviceToHost);

  return host_net;
}

__host__ void net_free_on_device(Net* device_net) {
  // Retrieve the device pointers for data
  Wire* device_bags;
  u32*  device_gidx;
  u32*  device_gtmp;
  Node* device_node;
  hipMemcpy(&device_bags, &(device_net->bags), sizeof(Wire*), hipMemcpyDeviceToHost);
  hipMemcpy(&device_gidx, &(device_net->gidx), sizeof(u32*),  hipMemcpyDeviceToHost);
  hipMemcpy(&device_gtmp, &(device_net->gtmp), sizeof(u32*),  hipMemcpyDeviceToHost);
  hipMemcpy(&device_node, &(device_net->node), sizeof(Node*), hipMemcpyDeviceToHost);

  // Free the device memory
  hipFree(device_bags);
  hipFree(device_gidx);
  hipFree(device_gtmp);
  hipFree(device_node);
  hipFree(device_net);
}

__host__ void net_free_on_host(Net* host_net) {
  free(host_net->bags);
  free(host_net->gidx);
  free(host_net->gtmp);
  free(host_net->node);
  free(host_net);
}

