#include "hip/hip_runtime.h"
#include <stdint.h>

typedef uint8_t u8;
typedef uint32_t u32;
typedef uint64_t u64;

// Pointers = 4-bit tag + 28-bit val
typedef u32 Ptr;

// Nodes are pairs of pointers
typedef struct alignas(8) {
  Ptr ports[2];
} Node;

// Wires are pairs of pointers
typedef struct alignas(8) {
  Ptr lft;
  Ptr rgt;
} Wire;

// An interaction net
typedef struct {
  Ptr   root; // root wire
  u32   blen; // total bag length (redex count)
  Wire* bags; // redex bags (active pairs)
  Node* node; // memory buffer with all nodes
  u32*  gidx; // aux buffer used on scatter fns
  Wire* gmov; // aux buffer used on scatter fns
  u32   pbks; // last blocks count used
  u32   done; // number of completed threads
  u32   rwts; // number of rewrites performed
} Net;

extern "C" __global__ void test(Net net) {
    assert(net.root == 0);
    assert(net.blet == 0);
    assert(net.pbks == 0);
    assert(net.done == 0);
    assert(net.rwts == 0);
}