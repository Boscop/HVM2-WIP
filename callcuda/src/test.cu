#include "hip/hip_runtime.h"
// #include <stdint.h>
typedef unsigned char      uint8_t;
typedef unsigned short     uint16_t;
typedef unsigned int       uint32_t;
typedef unsigned long long uint64_t;

typedef uint8_t u8;
typedef uint32_t u32;
typedef uint64_t u64;

// Configuration
// -------------

// This code is initially optimized for nVidia RTX 4090
const u32 BLOCK_LOG2    = 8;                                     // log2 of block size
const u32 BLOCK_SIZE    = 1 << BLOCK_LOG2;                       // threads per block
const u32 UNIT_SIZE     = 4;                                     // threads per rewrite unit
const u32 NODE_SIZE     = 1 << 28;                               // max total nodes (2GB addressable)
const u32 BAGS_SIZE     = BLOCK_SIZE * BLOCK_SIZE * BLOCK_SIZE;  // size of global redex bag
const u32 GROUP_SIZE    = BLOCK_SIZE * BLOCK_SIZE;               // size os a group of bags
const u32 GIDX_SIZE     = BAGS_SIZE + GROUP_SIZE + BLOCK_SIZE;   // aux object to hold scatter indices
const u32 GMOV_SIZE     = BAGS_SIZE;                             // aux object to hold scatter indices

// Pointers = 4-bit tag + 28-bit val
typedef u32 Ptr;

// Nodes are pairs of pointers
typedef struct alignas(8) {
  Ptr ports[2];
} Node;

// Wires are pairs of pointers
typedef struct alignas(8) {
  Ptr lft;
  Ptr rgt;
} Wire;

// An interaction net
typedef struct {
  Ptr   root; // root wire
  u32   blen; // total bag length (redex count)
  Wire* bags; // redex bags (active pairs)
  Node* node; // memory buffer with all nodes
  u32*  gidx; // aux buffer used on scatter fns
  Wire* gmov; // aux buffer used on scatter fns
  u32   pbks; // last blocks count used
  u32   done; // number of completed threads
  u32   rwts; // number of rewrites performed
} Net;

extern "C" __global__ void test(Net* net) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id == 0) {
        return;
    }

    assert(net->root == 0);
    assert(net->blen == 0);
    assert(net->pbks == 0);
    assert(net->done == 0);
    assert(net->rwts == 0);

    assert(net->bags != NULL);
    assert(net->node != NULL);
    assert(net->gidx != NULL);
    assert(net->gmov != NULL);

    assert(net->bags[0].lft == 10);
    assert(net->bags[0].rgt == 20);
    for (u32 i = 1; i < BAGS_SIZE; ++i) {
        assert(net->bags[i].lft == 0);
        assert(net->bags[i].rgt == 0);
    }
    /* for (u32 i = 0; i < NODE_SIZE; ++i) {
        assert(net->node->ports[0] == 0);
        assert(net->node->ports[1] == 0);
    }
    for (u32 i = 0; i < GIDX_SIZE; ++i) {
        assert(net->gidx[i] == 0);
    }
    for (u32 i = 0; i < GMOV_SIZE; ++i) {
        assert(net->gmov[i].lft == 0);
        assert(net->gmov[i].rgt == 0);
    } */
}